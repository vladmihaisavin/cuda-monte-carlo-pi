#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <ctime>

#define NUM_BLOCKS 1024
#define NUM_THREADS_PER_BLOCK 1024
#define NUM_SAMPLES 1e6

float* generateSamples(int size) {
    float* array = new float[size];
    srand(time(NULL));

    for (int i = 0; i < size; ++i) {
        array[i] = (float) rand() / RAND_MAX;
    }
    return array;
}

float getPiValue(int numSamplesInCircle) {
    return 4.0 * (float) numSamplesInCircle / NUM_SAMPLES;
}

hipError_t countSamplesInCircleWithCuda(float* host_randX, float* host_randY, int* host_sampleCountPerBlock);

__global__ void countSamplesInCircleKernel(float* device_randX, float* device_randY, int* device_sampleCountPerBlock)
{
    __shared__ int sampleCountPerThreads[NUM_THREADS_PER_BLOCK];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int samplesInCircle = 0;

    for (int i = idx; i < NUM_SAMPLES; i += blockDim.x * NUM_BLOCKS) {
        if (device_randX[i] * device_randX[i] + device_randY[i] * device_randY[i] <= 1.0f) {
            samplesInCircle++;
        }
    }

    sampleCountPerThreads[threadIdx.x] = samplesInCircle;

    if (threadIdx.x == 0) {
        int totalSamplesInCircle = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            totalSamplesInCircle += sampleCountPerThreads[i];
        }
        device_sampleCountPerBlock[blockIdx.x] = totalSamplesInCircle;
    }
}

int main()
{
    std::clock_t c_start = std::clock();
    printf("Generating host X samples...\n");
    float* host_randX = generateSamples(NUM_SAMPLES);

    printf("Generating host Y samples...\n");
    float* host_randY = generateSamples(NUM_SAMPLES);
    std::clock_t c_end = std::clock();

    long double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    printf("Generated vectors in: %lf ms\n", time_elapsed_ms);

    int* host_sampleCountPerBlock = new int[NUM_BLOCKS];

    // Call kernel    
    hipError_t cudaStatus = countSamplesInCircleWithCuda(host_randX, host_randY, host_sampleCountPerBlock);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "countSamplesInCircleWithCuda failed!");
        return 1;
    }

    int numSamplesInCircle = 0;
    for (int i = 0; i < NUM_BLOCKS; ++i) {
        numSamplesInCircle += host_sampleCountPerBlock[i];
    }

    float piValue = getPiValue(numSamplesInCircle);
    printf("pi = %f\n", piValue);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t countSamplesInCircleWithCuda(float* host_randX, float* host_randY, int* host_sampleCountPerBlock)
{
    hipError_t cudaStatus;
    hipEvent_t start1, stop1, start2, stop2, start3, stop3;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventCreate(&start3);
    hipEventCreate(&stop3);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    const size_t SAMPLE_SIZE = NUM_SAMPLES * sizeof(float);
    float* device_randX;
    float* device_randY;

    printf("Device malloc randX...\n");
    hipEventRecord(start1, 0);
    cudaStatus = hipMalloc(&device_randX, SAMPLE_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (device_randX)\n");
        goto Error;
    }

    printf("Device malloc randY...\n");
    cudaStatus = hipMalloc(&device_randY, SAMPLE_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (device_randY)\n");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    printf("Copying randX from host to device...\n");
    cudaStatus = hipMemcpy(device_randX, host_randX, SAMPLE_SIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (device_randX)\n");
        goto Error;
    }

    printf("Copying randY from host to device...\n");
    cudaStatus = hipMemcpy(device_randY, host_randY, SAMPLE_SIZE, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (device_randY)\n");
        goto Error;
    }
    hipEventRecord(stop1, 0);
    hipEventSynchronize(stop1);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start1, stop1);
    printf("Malloc and copy from --proc-- to --graphics card-- time: %f\n", elapsedTime);

    const size_t COUNT_SIZE = NUM_BLOCKS * sizeof(int);

    hipEventRecord(start2, 0);
    printf("Device malloc sampleCountPerBlock...\n");
    int* device_sampleCountPerBlock;
    cudaStatus = hipMalloc(&device_sampleCountPerBlock, COUNT_SIZE);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (device_sampleCountPerBlock)\n");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    printf("Launching kernel...\n");
    countSamplesInCircleKernel<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(device_randX, device_randY, device_sampleCountPerBlock);

    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&elapsedTime, start2, stop2);
    printf("Kernel execution time: %f\n", elapsedTime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    
    hipEventRecord(start3, 0);
    printf("Copying sampleCountPerBlock from device to host...\n");
    cudaStatus = hipMemcpy(host_sampleCountPerBlock, device_sampleCountPerBlock, COUNT_SIZE, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (device_sampleCountPerBlock)\n");
        goto Error;
    }
    hipEventRecord(stop3, 0);
    hipEventSynchronize(stop3);
    hipEventElapsedTime(&elapsedTime, start3, stop3);
    printf("Copy from --graphics card-- to --proc-- time: %f\n", elapsedTime);

Error:
    hipFree(device_randX);
    hipFree(device_randY);
    hipFree(device_sampleCountPerBlock);
    
    return cudaStatus;
}
